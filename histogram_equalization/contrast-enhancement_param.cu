#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include "hist-equ.h"
#include <time.h>

#define cudaCheckError() {                                                                       \
        hipError_t e=hipGetLastError();                                                        \
        if(e!=hipSuccess) {                                                                     \
            printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));\
            hipFree(d_Histogram);\
            hipFree(d_Input);\
            hipHostFree(result.img);\
            hipFree(blocked_hists);\
            hipHostFree(img_in.img);\
            hipDeviceReset();\
            exit(EXIT_FAILURE);                                                                  \
        }                                                                                        \
    }

PGM_IMG contrast_enhancement_g(PGM_IMG img_in)
{
    PGM_IMG result;
    int hist[256];
    
    int h_Histogram[256];//histogram gia thn gpu sto host
    
    int *d_Histogram = NULL;
    unsigned char *d_Input = NULL;
    unsigned int img_size = img_in.w * img_in.h;
    int *blocked_hists;
    
    int thread_reps;
    int block_num;
    int size_in_blocks = img_size / 256;//apo poses 256ades apoteleitai h eikona
    int which_kernel;
    
    int j = 0;
    int cdf = 0;
    int min = 0;
    int d = 0;
    int lut[256];
    
    int gpu_calc;//poso meros ths eikonas tha ypologisei h gpu
    
    //gia tis metriseis
    
    hipStream_t stream;
    hipEvent_t start_hist,end_hist;
    float hist_time;
    
    hipEventCreate(&start_hist);
    hipEventCreate(&end_hist);
    
    result.w = img_in.w;
    result.h = img_in.h;

    hipHostAlloc((void **) &result.img, result.w * result.h * sizeof(unsigned char), hipHostMallocDefault);
    
    printf("Number of blocks(256) of the image is: %d\n", size_in_blocks);
    printf("Enter how many blocks the gpu will compute : ");
    scanf("%d",&gpu_calc);
    
    if(gpu_calc > size_in_blocks){
        
        printf("Number of blocks should be less or equal than %d\n",gpu_calc);
        
        hipHostFree(result.img);
        hipHostFree(img_in.img);
        
        hipDeviceReset();
        
        exit(1);
    }
    
    printf("Number of blocks for GPU is: %d and block size is 256...\n",gpu_calc);
    printf("Enter how many reps a thread will do (Enter a power of two): ");
    scanf("%d",&thread_reps);
    
    printf("Enter which kernel to run:(1-5) ");
    scanf("%d",&which_kernel);
    
    if(which_kernel > 5){
        
        printf("Pick a proper kernel no (1-5)\n");
        
        hipHostFree(result.img);
        hipHostFree(img_in.img);
        
        hipDeviceReset();
        
        exit(1);
    }
    
    block_num = gpu_calc / thread_reps;
    
    if(block_num == 0){
        
        printf("Too few blocks or too many thread reps\n");
        
        hipHostFree(result.img);
        hipHostFree(img_in.img);
        
        hipDeviceReset();
        
        exit(1);
    }
    
    hipMalloc((void**)&d_Histogram,sizeof(int) * 256);
    hipMalloc((void**)&d_Input,sizeof(unsigned char) * gpu_calc * 256);//na to allakso
    hipMalloc((void**)&blocked_hists,sizeof(int) * 256 * block_num);
    
    if(d_Histogram == NULL || d_Input == NULL || blocked_hists == NULL){
        
        printf("Error in device memory allocation.Exiting...\n");
        
        hipHostFree(result.img);
        hipHostFree(img_in.img);
                
        hipDeviceReset();
        exit(1);
    }

    //hipMemset(blocked_hists, 0, sizeof(int) * 256 * block_num);
    hipStreamCreate(&stream);

    hipEventRecord(start_hist,stream);        
   
    
    hipMemset(d_Histogram,0,256 * sizeof(int));
    
    cudaCheckError();
    
    //metafora eikonas
    
    hipMemcpyAsync(d_Input, img_in.img ,gpu_calc * 256 * sizeof(unsigned char), hipMemcpyHostToDevice,stream);//async meta

    cudaCheckError();
    
    //kernels
    
    if(which_kernel == 1){
        
        histogram_g1<<<block_num , 256, 0, stream>>>(d_Histogram ,d_Input,gpu_calc * 256);
        
        cudaCheckError();
        
    }else if(which_kernel == 2){

        hipMemset(blocked_hists, 0, sizeof(int) * 256 * block_num);

        cudaCheckError();

        histogram_g2<<<block_num , 256, 0, stream>>>(d_Histogram , d_Input,blocked_hists,gpu_calc * 256);

        cudaCheckError();
        
    }else if(which_kernel == 3){
        
        histogram_s1<<<block_num , 256, 0, stream>>>(d_Histogram ,d_Input,gpu_calc * 256);
        
        cudaCheckError();
    
    }else if(which_kernel == 4){
        
        histogram_s3<<<block_num , 256, 0, stream>>>(d_Histogram ,d_Input,gpu_calc * 256);
        
        cudaCheckError();

    }else if(which_kernel == 5){
        
        hipMemset(blocked_hists, 0, sizeof(int) * 256 * block_num);

        cudaCheckError();

        histogram_s4<<<block_num , 256, 0, stream>>>(d_Histogram ,d_Input,blocked_hists, gpu_calc * 256);

        cudaCheckError();
    
    }
    
    hipMemcpyAsync(h_Histogram,d_Histogram,256* sizeof(int),hipMemcpyDeviceToHost,stream);
    
    cudaCheckError();
    

    for(int i = 0; i < 256; i++)//isos na figei meta
        hist[i] = 0;
    
    //cpu computation
    
    for(int i = gpu_calc * 256; i < img_size; i++)
        hist[img_in.img[i]]++;
    
    hipDeviceSynchronize();
    
    //merge gpu and cpu
    
    for(int i = 0; i < 256; i++)
        hist[i] += h_Histogram[i];
    
    /* ypologismos cdf sth cpu*/
    
    while(min == 0){
        
        lut[j] = 0;
        min = hist[j];
        
        j++;
    }
    
    d = img_in.w *img_in.h - min;
    j--;
    
    for(; j < 256; j++){
        
        cdf += hist[j];

        lut[j] = (int)(((float)cdf - min)*255/d + 0.5);
        
        if(lut[j] > 255)
            lut[j] = 255;
        
        if(lut[j] < 0)
            lut[j] = 0;
    }
    
    hipMemcpyAsync(d_Histogram,lut,256 * sizeof(int),hipMemcpyHostToDevice,stream);//metaferontai sthn d_Histogram gia na mhn ksanadesmeyseis mnhmh
    
    cudaCheckError();
    
    result_calc_param<<<gpu_calc , 256, 0, stream>>>(d_Input,d_Histogram);//d_Result grafetai pano sto d_Input
    
    cudaCheckError();
    
    hipMemcpyAsync(result.img,d_Input, 256 * gpu_calc * sizeof(unsigned char),hipMemcpyDeviceToHost,stream);
    
    cudaCheckError();
    
    for(int i = gpu_calc * 256; i < img_size; i ++)
        result.img[i] = (unsigned char)lut[img_in.img[i]];
    
    hipEventRecord(end_hist,stream);
    
    hipEventSynchronize(end_hist);
    
    hipEventElapsedTime(&hist_time,start_hist,end_hist);
    
    cudaCheckError();

    fprintf(stderr,"Total time of histogram(%d) = %lf\n",which_kernel,hist_time/1000);
	    
    hipFree(d_Histogram);
    hipFree(d_Input);
    hipFree(blocked_hists);
    
    hipEventDestroy(start_hist);
    hipEventDestroy(end_hist);
    
    hipStreamDestroy(stream);
    
    return result;
}
