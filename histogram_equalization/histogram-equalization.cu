#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include "hist-equ.h"

//TO PIO APLO:
//KATHE THREAD DIAVAZEI ENA(h kai perissotera) STOIXEIO KAI GRAFEI TO APOTELESMA STO ANTISTOIXO STOIXEIO TOU global_histogram

__global__ void histogram_g1(int * hist_out , unsigned char * img_in, int size_of_img){
    
    int index = threadIdx.x + blockDim.x * blockIdx.x;//deixnei poio stoixeio tha parei apth mnhmh
    
    for(; index < size_of_img; index += gridDim.x * blockDim.x)
        atomicAdd(&hist_out[img_in[index]],1);
}

//KATHE THREAD DIAVAZEI ENA STOIXEIO(h kai perissotera) KAI GRAFEI TO APOTELESMA STO block_histogram.META KATHE THREAD PAIRNEI TO ANTISTOIXO STOIXEIO APTO
//block_histogram KAI TO APOTHIKEYEI STO global_histogram

__global__ void histogram_g2(int * hist_out, unsigned char * img_in,int *block_histogram,int size_of_img){
    
    int index = threadIdx.x + blockDim.x * blockIdx.x;//deixnei poio stoixeio tha parei apth mnhmh
    int current_block_histogram = blockIdx.x * 256;//block_histogram einai pinakas 256 * num_of_blocks.opote current_block_histogram deixnei sto antistoixo block
    
    for(; index < size_of_img; index += gridDim.x * blockDim.x)
        atomicAdd(&block_histogram[current_block_histogram + img_in[index]],1);
    
    __syncthreads();
    
    if(block_histogram[current_block_histogram + threadIdx.x])
        atomicAdd(&hist_out[threadIdx.x],block_histogram[current_block_histogram + threadIdx.x]);//na to ksanado
}

/*TA IDIA ALLA ME SHARED*/

__global__ void histogram_s1(int * hist_out, unsigned char * img_in, int size_of_img){
    
    int index = threadIdx.x + blockDim.x * blockIdx.x;//deixnei poio stoixeio tha parei apth mnhmh
    
    __shared__ int block_histogram[256];
    
    //kane 0 to block_histogram
    block_histogram[threadIdx.x] = 0;
    
    __syncthreads();
    
    for(; index < size_of_img; index += gridDim.x * blockDim.x)
        atomicAdd(&block_histogram[img_in[index]],1);
    
    __syncthreads();
    
    if(block_histogram[threadIdx.x])
       atomicAdd(&hist_out[threadIdx.x],block_histogram[threadIdx.x]);//na to ksanado
}

/*warp - aggregation*/

__global__ void histogram_s3(int * hist_out , unsigned char * img_in , int size_of_img){
    
    __shared__ int block_histogram[256];
    
    unsigned int peers = 0;
    short int is_peer = 0;
    
    //All lanes are available
    unsigned int unclaimed = 0xffffffff;//set all 32 bits
    
    int index = threadIdx.x + blockDim.x * blockIdx.x;//deixnei poio stoixeio tha parei apth mnhmh
    int element_of_in;
    int lane_num = threadIdx.x % 32;//arithos tou thread mesa sto warp
    
    block_histogram[threadIdx.x] = 0;
    
    __syncthreads();
    
    for(; index < size_of_img; index += gridDim.x * blockDim.x){
        
        element_of_in = img_in[index];
        
        do {
            // fetch key of first unclaimed lane and compare with this key
            is_peer = (element_of_in == __shfl_sync(unclaimed,element_of_in, __ffs(unclaimed) - 1));

            // determine which lanes had a match
            peers = __ballot_sync(unclaimed,is_peer);

            // remove lanes with matching keys from the pool
            unclaimed ^= peers;

        // quit if we had a match
        }while (!is_peer);
        
        if(lane_num == __ffs(peers) - 1)
            atomicAdd(&block_histogram[element_of_in], __popc(peers)); 
        
        unclaimed = 0xffffffff;
        
        __syncwarp();
    }
    
    __syncthreads();
    
    if(block_histogram[threadIdx.x])
       atomicAdd(&hist_out[threadIdx.x],block_histogram[threadIdx.x]);
}


//kai vale memset mesa sto if

__global__ void histogram_s4(int * hist_out , unsigned char * img_in ,int * block_histogram, int size_of_img){
    
    int current_block_histogram = blockIdx.x * 256;
    
    unsigned int peers = 0;
    short int is_peer = 0;
    
    //All lanes are available
    unsigned int unclaimed = 0xffffffff;//set all 32 bits
    
    int index = threadIdx.x + blockDim.x * blockIdx.x;//deixnei poio stoixeio tha parei apth mnhmh
    int element_of_in;
    int lane_num = threadIdx.x % 32;//arithos tou thread mesa sto warp
    
    for(; index < size_of_img; index += gridDim.x * blockDim.x){
        
        element_of_in = img_in[index];
        
        do {
            // fetch key of first unclaimed lane and compare with this key
            is_peer = (element_of_in == __shfl_sync(unclaimed,element_of_in, __ffs(unclaimed) - 1));

            // determine which lanes had a match
            peers = __ballot_sync(unclaimed,is_peer);

            // remove lanes with matching keys from the pool
            unclaimed ^= peers;

        // quit if we had a match
        }while (!is_peer);
        
        if(lane_num == __ffs(peers) - 1)
            atomicAdd(&block_histogram[current_block_histogram + element_of_in],__popc(peers));
        
        unclaimed = 0xffffffff;
        
        __syncwarp();
    }
    
    __syncthreads();
    
    if(block_histogram[current_block_histogram + threadIdx.x])
       atomicAdd(&hist_out[threadIdx.x],block_histogram[current_block_histogram + threadIdx.x]);
} 


__global__ void result_calc(unsigned char * img_out,int * lut,unsigned int img_size){//d_Input kai result vriskontai sthn idia mnhmh gia eksikonomisi xoroy

    __shared__ int shared_lut[256];
    int index =  threadIdx.x + blockDim.x * blockIdx.x;
    
    shared_lut[threadIdx.x] = lut[threadIdx.x];

    __syncthreads();
    
    for(; index < img_size; index += gridDim.x * blockDim.x)
        img_out[index] = (unsigned char)shared_lut[img_out[index]];//sigoureyoume sth cpu oti 0 <= lut[img_in[index]] <= 255 
}

__global__ void result_calc_param(unsigned char * img_out,int * lut){//d_Input kai result vriskontai sthn idia mnhmh gia eksikonomisi xoroy

    __shared__ int shared_lut[256];

    shared_lut[threadIdx.x] = lut[threadIdx.x];

    __syncthreads();

    int index =  threadIdx.x + blockDim.x * blockIdx.x;
    
    img_out[index] = (unsigned char)shared_lut[img_out[index]];//sigoureyoume sth cpu oti 0 <= lut[img_in[index]] <= 255 
}
