#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include "hist-equ.h"
#include <time.h>

#define cudaCheckError() {                                                                       \
        hipError_t e=hipGetLastError();                                                        \
        if(e!=hipSuccess) {                                                                     \
            printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));\
            hipFree(d_Histogram);\
            hipFree(d_Input);\
            hipHostFree(result.img);\
            hipFree(blocked_hists);\
            hipHostFree(img_in.img);\
            hipDeviceReset();\
            exit(EXIT_FAILURE);                                                                  \
        }                                                                                        \
    }

PGM_IMG contrast_enhancement_g(PGM_IMG img_in)
{
    PGM_IMG result;
    int hist[256];
    unsigned int img_size = img_in.w * img_in.h;
    
    int *d_Histogram = NULL;
    unsigned char *d_Input = NULL;
    int *blocked_hists = NULL;
    
    int block_num;
    int size_in_blocks = img_size / 256;//apo poses 256ades apoteleitai h eikona
    
    int j = 0;
    int cdf = 0;
    int min = 0;
    int d = 0;
    int lut[256];
    

    
    hipStream_t stream;
    hipEvent_t start_hist,end_hist;
    float hist_time;
    

    
    hipEventCreate(&start_hist);
    hipEventCreate(&end_hist);
    
    result.w = img_in.w;
    result.h = img_in.h;

    hipHostAlloc((void **) &result.img, img_size * sizeof(unsigned char), hipHostMallocDefault);

   
    block_num = size_in_blocks / 32;//32 reps(h 33...analoga to ypoloipo) kathe thread
    
    if(block_num == 0){
        
        printf("Image too small\n");
        
        exit(1);
    }
    
    hipMalloc((void**)&d_Histogram,sizeof(int) * 256);
    hipMalloc((void**)&d_Input,sizeof(unsigned char) * img_size);//olo sth gpu?
    hipMalloc((void**)&blocked_hists,sizeof(int) * 256 * block_num);
    
    if(d_Histogram == NULL || d_Input == NULL || blocked_hists == NULL){
        
        printf("Error in device memory allocation.Exiting...\n");
        hipDeviceReset();
        exit(1);
    }

    hipStreamCreate(&stream);

    hipEventRecord(start_hist,stream);        
    
    hipMemsetAsync(d_Histogram,0,256 * sizeof(int));
    
    cudaCheckError();
    
    hipMemsetAsync(blocked_hists, 0, sizeof(int) * 256 * block_num);

    cudaCheckError();
    
    //metafora eikonas
    
    hipMemcpyAsync(d_Input, img_in.img ,img_size * sizeof(unsigned char), hipMemcpyHostToDevice,stream);

    cudaCheckError();
    
    //kernel
    
    histogram_g2<<<block_num , 256, 0, stream>>>(d_Histogram , d_Input,blocked_hists,img_size);

    cudaCheckError();
    
    //metafora histogram sth cpu
    
    hipMemcpyAsync(hist,d_Histogram,256* sizeof(int),hipMemcpyDeviceToHost,stream);
    
    cudaCheckError();
    
    hipDeviceSynchronize();
        
    /* ypologismos cdf */
    
    while(min == 0){
        
        lut[j] = 0;
        min = hist[j];
        
        j++;
    }
    
    d = img_size - min;
    j--;
    
    for(; j < 256; j++){
        
        cdf += hist[j];

        lut[j] = (int)(((float)cdf - min)*255/d + 0.5);
        
        if(lut[j] > 255)
            lut[j] = 255;
        
        if(lut[j] < 0)
            lut[j] = 0;
    }
    
    hipMemcpyAsync(d_Histogram,lut,256 * sizeof(int),hipMemcpyHostToDevice,stream);//metaferontai sthn d_Histogram gia na mhn ksanadesmeyseis mnhmh
    
    cudaCheckError();
    
    result_calc<<<size_in_blocks / 32, 256, 0, stream>>>(d_Input,d_Histogram,img_size);
    
    cudaCheckError();
    
    hipMemcpyAsync(result.img,d_Input, img_size,hipMemcpyDeviceToHost,stream);
    
    cudaCheckError();
    
    hipEventRecord(end_hist,stream);
    
    hipEventSynchronize(end_hist);
    
    hipEventElapsedTime(&hist_time,start_hist,end_hist);
    
    cudaCheckError();

    fprintf(stderr,"Total time = %lf\n",hist_time/1000);
	    
    hipFree(d_Histogram);
    hipFree(d_Input);
    hipFree(blocked_hists);
    
    hipEventDestroy(start_hist);
    hipEventDestroy(end_hist);
    
    hipStreamDestroy(stream);
    
    return result;
}
